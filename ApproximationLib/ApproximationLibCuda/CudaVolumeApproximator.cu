#include "hip/hip_runtime.h"
#include <ApproximationLibCuda/CudaVolumeApproximator.h>

#include <hip/hip_runtime.h>
#include <>

#include <thrust/copy.h>
#include <thrust/device_make_unique.h>
#include <thrust/device_vector.h>
#include <thrust/memory.h>
#include <thrust/transform.h>

#include <memory>

#include <fmt/core.h>



namespace
{
__global__ void ComputePoint(const thrust::device_ptr<const float> triangles,
    const std::size_t trianglesSize,
    const thrust::device_ptr<const float> points,
    const std::size_t pointsSize,
    const thrust::device_ptr<const float> minExtent,
    const thrust::device_ptr<const float> maxExtent)
{
	printf("Triangles size: %f, points size: %f, minExtent: {%f,%f,%f}, maxExtent: {%f,%f,%f}\n",
	    static_cast<float>(trianglesSize), static_cast<float>(pointsSize),
	    static_cast<float>(minExtent[0]), static_cast<float>(minExtent[1]),
	    static_cast<float>(minExtent[2]), static_cast<float>(maxExtent[0]),
	    static_cast<float>(maxExtent[1]), static_cast<float>(maxExtent[2]));
}
} // namespace

namespace VolumeApproximation::Impl
{

bool CudaVolumeApproximator::IsAvialiable() const noexcept
{
	// Should be avialiable if we have at least one device
	int devicesCount{};
	if (hipGetDeviceCount(&devicesCount) != hipError_t::hipSuccess)
	{
		return false;
	}
	return devicesCount > 0;
}

ApproximationResult CudaVolumeApproximator::ComputeVolume(const std::vector<Triangle>& triangles,
    const std::vector<Vector3F>& randomPoints,
    const Vector3F& minimumExtent,
    const Vector3F& maximumExtent)
{
	// const thrust::device_vector<float> gpuTriangles{triangles};
	// const thrust::device_vector<float> gpuPoints{randomPoints};
	// const thrust::device_vector<float> gpuMinExtent{minimumExtent, minimumExtent + 3};
	// const thrust::device_vector<float> gpuMaxExtent{maximumExtent, maximumExtent + 3};

	// ComputePoint<<<1, 1>>>(gpuTriangles.data(), gpuTriangles.size(), gpuPoints.data(),
	//     gpuPoints.size(), gpuMinExtent.data(), gpuMaxExtent.data());

	return {};
}
} // namespace VolumeApproximation::Impl