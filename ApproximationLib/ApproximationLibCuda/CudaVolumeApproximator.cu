#include "hip/hip_runtime.h"
#include <ApproximationLibCuda/CudaVolumeApproximator.h>

#include <hip/hip_runtime.h>
#include <>

#include <thrust/copy.h>
#include <thrust/device_make_unique.h>
#include <thrust/device_vector.h>
#include <thrust/memory.h>
#include <thrust/transform.h>

#include <memory>

#include <fmt/core.h>



namespace
{
__global__ void ComputePoint(const thrust::device_ptr<const float> triangles,
    const std::size_t trianglesSize,
    const thrust::device_ptr<const float> points,
    const std::size_t pointsSize,
    const thrust::device_ptr<const float> minExtent,
    const thrust::device_ptr<const float> maxExtent)
{
	printf("Triangles size: %f, points size: %f, minExtent: {%f,%f,%f}, maxExtent: {%f,%f,%f}\n",
	    static_cast<float>(trianglesSize), static_cast<float>(pointsSize),
	    static_cast<float>(minExtent[0]), static_cast<float>(minExtent[1]),
	    static_cast<float>(minExtent[2]), static_cast<float>(maxExtent[0]),
	    static_cast<float>(maxExtent[1]), static_cast<float>(maxExtent[2]));
}
} // namespace

namespace VolumeApproximation::Impl
{

bool CudaVolumeApproximator::IsAvialiable() const
{
	// Should be avialiable if we have at least one device
	int devicesCount{};
	if (hipGetDeviceCount(&devicesCount) != hipError_t::hipSuccess)
	{
		return false;
	}
	return devicesCount > 0;
}

ApproximationResult CudaVolumeApproximator::ComputeVolume(const std::vector<Triangle>& triangles,
    const std::vector<Vector3F>& randomPoints,
    const Vector3F& minimumExtent,
    const Vector3F& maximumExtent)
{
	// TODO: Implement the CUDA version
	return {};
}
} // namespace VolumeApproximation::Impl